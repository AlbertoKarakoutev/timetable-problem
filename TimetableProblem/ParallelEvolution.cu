
#include <hip/hip_runtime.h>
﻿/*
#include <cuda_runtime.h>
#include <vector>

#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>

#include <device_launch_parameters.h>

#include "Utilities.h"
#include "Lecture.h"
#include "gVariables.h"

struct ModelData {
    vector<vector<vector<Lecture>>> solution;
    vector<int> solutionScore = { -1000, -1000, -1000 };
    string modelID;
};

static __global__ void evolveAndEvaluate(const vector<vector<vector<Lecture>>>* solution, ModelData* modelData){
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    modelData[tid].solutionScore = Utilities::evaluate(modelData[tid].solution);
    modelData[tid].solution = Utilities::evolve(solution[tid]);
    modelData[tid].modelID = Utilities::identify(modelData[tid].solution);

}

vector<ModelData> generate(vector<vector<vector<Lecture>>> solutionInput)
{

    ModelData result;
    vector<vector<vector<Lecture>>> solution = solutionInput;

    int solutionSize = sizeof(solution);
    
    int solutionSize_max = daysInWeek * periodInDay * 2 * roomCount * (5 * sizeof(int) + 3 * sizeof(string) + studentsPerClass * sizeof(int));
    int scoreSize = 3 * sizeof(int);
    int idSize = sizeof(string);
    int resultSize = solutionSize_max + scoreSize + idSize;

    vector<vector<vector<Lecture>>>* d_solution;
    ModelData* d_result;

    cudaMalloc(&d_solution, solutionSize);
    cudaMalloc(&d_result, resultSize);

    cudaMemcpy(&d_solution, &solution, solutionSize, cudaMemcpyHostToDevice);
    cudaMemcpy(&d_result, &result, resultSize, cudaMemcpyHostToDevice);

    int NUM_THREADS = 1 << 10;
    int NUM_BLOCKS = (solutionSize_max + NUM_THREADS - 1) / NUM_THREADS;

    evolveAndEvaluate<<<NUM_BLOCKS, NUM_THREADS>>>(d_solution, d_result);

    cudaMemcpy(&result, d_result, resultSize, cudaMemcpyDeviceToHost);

    vector<ModelData> results;

    results.push_back(result);

    return results;
}

__global__ vector<int> evaluate(vector<vector<vector<Lecture>>> schedule)
{
    vector<vector<vector<Lecture>>> model = schedule;

    int repeat = 0;
    int coursesNotPreceding = 0;

    int coursesNotInRange = 0;

    for (int day_index = 0; day_index < model.size(); day_index++) {
        for (int period_index = 0; period_index < model[day_index].size(); period_index++) {
            repeat -= duplicateCount(model, day_index, period_index);
        }
    }

    for (int i = 0; i < cLength; i++) {
        Course course = *getCourse(i);
        if (courseInRange(model, course) != 0) {
            coursesNotInRange--;
        }
        for (string precedingCourse : course.getPrecedingCourseNames()) {
            if (precedingCourse.compare("") != 0) {
                if (!courseIsPreceeded(model, course.getName(), precedingCourse)) {
                    coursesNotPreceding--;
                }
            }
        }
    }

    vector<int> score = { repeat, coursesNotInRange, coursesNotPreceding };
    return score;
}

__global__ int duplicateCount(vector<vector<vector<Lecture>>> model, int targetDay, int targetPeriod)
{
    int hits = 0;

    vector<Lecture> period = model.at(targetDay).at(targetPeriod);

    vector<string> periodTeachers;
    vector<int> periodRooms;
    vector<Course> periodCourses;
    vector<int> periodStudents;

    vector<string> checkedCoursesForStudents;

    for (Lecture lecture : period) {
        periodTeachers.push_back(lecture.getTeacher());
        periodRooms.push_back(lecture.getRoom());
        periodCourses.push_back(lecture.getCourse());
        int studentsSize = static_cast<int>(lecture.getCourse().getStudents().size());
        if (count(checkedCoursesForStudents.begin(), checkedCoursesForStudents.end(), lecture.getCourse().getName()) < 1) {
            vector<int> lStudents = lecture.getCourse().getStudents();
            periodStudents.insert(periodStudents.end(), lStudents.begin(), lStudents.end());
            checkedCoursesForStudents.push_back(lecture.getCourse().getName());
        }
    }

    vector<string> checkedTeachers;
    for (string teacher : periodTeachers) {
        if (count(checkedTeachers.begin(), checkedTeachers.end(), teacher) > 0)
            continue;
        if (count(periodTeachers.begin(), periodTeachers.end(), teacher) > 1) {
            checkedTeachers.push_back(teacher);
            hits++;
        }
    }

    vector<int> checkedRooms;
    for (int room : periodRooms) {
        if (count(checkedRooms.begin(), checkedRooms.end(), room) > 0)
            continue;
        if (count(periodRooms.begin(), periodRooms.end(), room) > 1) {
            checkedRooms.push_back(room);
            hits++;
        }
    }

    vector<Course> checkedCourses;
    for (int i = 0; i < periodCourses.size(); i++) {
        Course course = periodCourses.at(i);
        if (count(checkedCourses.begin(), checkedCourses.end(), course) > 0)
            continue;
        if (count(periodCourses.begin(), periodCourses.end(), course) > 1) {
            checkedCourses.push_back(course);
            hits++;
        }
    }

    vector<int> checkedStudents;
    for (int student : periodStudents) {
        if (count(checkedStudents.begin(), checkedStudents.end(), student) > 0)
            continue;
        if (count(periodStudents.begin(), periodStudents.end(), student) > 1) {
            checkedStudents.push_back(student);
            hits++;
        }
    }

    return hits;
}

__global__ int courseInRange(vector<vector<vector<Lecture>>> model, Course course)
{
    int hits = 0;

    for (vector<vector<Lecture>> day : model) {
        for (vector<Lecture> period : day) {
            for (Lecture lecture : period) {
                if (lecture.getCourse() == course) {
                    hits++;
                }
            }
        }
    }
    if (hits < course.getMinPeriods()) {
        return -1;
    }
    if (hits > course.getMaxPeriods()) {
        return 1;
    }
    return 0;
}

__global__ bool courseIsPreceeded(vector<vector<vector<Lecture>>> schedule, string courseName, string precedingCourseName)
{
    vector<vector<vector<Lecture>>> model = schedule;

    int courseEarliestOccurence[2] = { static_cast<int>(model.size()) + 1, static_cast<int>(model.at(0).size()) + 1 };
    int preceedingCourseEarliestOccurence[2] = { static_cast<int>(model.size()), static_cast<int>(model.at(0).size()) };

    for (int day = 0; day < model.size(); day++) {
        for (int period = 0; period < model[day].size(); period++) {
            for (int lecture = 0; lecture < model[day][period].size(); lecture++) {
                if (model[day][period][lecture].getCourse().getName().compare(
                        courseName)
                    == 0) {
                    courseEarliestOccurence[0] = day;
                    courseEarliestOccurence[1] = period;
                    break;
                }
            }
        }
    }

    for (int day = 0; day < model.size(); day++) {
        for (int period = 0; period < model[day].size(); period++) {
            for (int lecture = 0; lecture < model[day][period].size(); lecture++) {
                if (model[day][period][lecture].getCourse().getName().compare(
                        precedingCourseName)
                    == 0) {
                    preceedingCourseEarliestOccurence[0] = day;
                    preceedingCourseEarliestOccurence[1] = period;
                    break;
                }
            }
        }
    }

    if (courseEarliestOccurence[0] != model.size() + 1 && courseEarliestOccurence[1] != model.at(0).size() + 1) {
        if (courseEarliestOccurence[0] > preceedingCourseEarliestOccurence[0]) {
            return true;
        } else {
            if (courseEarliestOccurence[1] > preceedingCourseEarliestOccurence[1]) {
                return true;
            }
        }
    }
    return false;
}

*/